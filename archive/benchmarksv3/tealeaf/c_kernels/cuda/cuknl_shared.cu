#include "../../shared.h"
#include "cuknl_shared.h"

void check_errors(int line_num, const char* file)
{
    hipDeviceSynchronize();

    int result = hipGetLastError();

    if(result != hipSuccess)
    {
        die(line_num, file, "Error in %s - return code %d (%s)\n",
                file, result, cuda_codes(result));
    }
}

// Enumeration for the set of potential CUDA error codes.
const char* cuda_codes(int code)
{
	switch(code)
	{
		case hipSuccess: return "hipSuccess"; // 0
		case hipErrorMissingConfiguration: return "hipErrorMissingConfiguration"; // 1
		case hipErrorOutOfMemory: return "hipErrorOutOfMemory"; // 2
		case hipErrorNotInitialized: return "hipErrorNotInitialized"; // 3
		case hipErrorLaunchFailure: return "hipErrorLaunchFailure"; // 4
		case hipErrorPriorLaunchFailure: return "hipErrorPriorLaunchFailure"; // 5
		case hipErrorLaunchTimeOut: return "hipErrorLaunchTimeOut"; // 6
		case hipErrorLaunchOutOfResources: return "hipErrorLaunchOutOfResources"; // 7
		case hipErrorInvalidDeviceFunction: return "hipErrorInvalidDeviceFunction"; // 8
		case hipErrorInvalidConfiguration: return "hipErrorInvalidConfiguration"; // 9
		case hipErrorInvalidDevice: return "hipErrorInvalidDevice"; // 10
		case hipErrorInvalidValue: return "hipErrorInvalidValue";// 11
		case hipErrorInvalidPitchValue: return "hipErrorInvalidPitchValue";// 12
		case hipErrorInvalidSymbol: return "hipErrorInvalidSymbol";// 13
		case hipErrorMapFailed: return "hipErrorMapFailed";// 14
		case hipErrorUnmapFailed: return "hipErrorUnmapFailed";// 15
		case cudaErrorInvalidHostPointer: return "cudaErrorInvalidHostPointer";// 16
		case hipErrorInvalidDevicePointer: return "hipErrorInvalidDevicePointer";// 17
		case hipErrorInvalidTexture: return "hipErrorInvalidTexture";// 18
		case cudaErrorInvalidTextureBinding: return "cudaErrorInvalidTextureBinding";// 19
		case hipErrorInvalidChannelDescriptor: return "hipErrorInvalidChannelDescriptor";// 20
		case hipErrorInvalidMemcpyDirection: return "hipErrorInvalidMemcpyDirection";// 21
		case cudaErrorAddressOfConstant: return "cudaErrorAddressOfConstant";// 22
		case cudaErrorTextureFetchFailed: return "cudaErrorTextureFetchFailed";// 23
		case cudaErrorTextureNotBound: return "cudaErrorTextureNotBound";// 24
		case cudaErrorSynchronizationError: return "cudaErrorSynchronizationError";// 25
		case cudaErrorInvalidFilterSetting: return "cudaErrorInvalidFilterSetting";// 26
		case cudaErrorInvalidNormSetting: return "cudaErrorInvalidNormSetting";// 27
		case cudaErrorMixedDeviceExecution: return "cudaErrorMixedDeviceExecution";// 28
		case hipErrorDeinitialized: return "hipErrorDeinitialized";// 29
		case hipErrorUnknown: return "hipErrorUnknown";// 30
		case cudaErrorNotYetImplemented: return "cudaErrorNotYetImplemented";// 31
		case cudaErrorMemoryValueTooLarge: return "cudaErrorMemoryValueTooLarge";// 32
		case hipErrorInvalidHandle: return "hipErrorInvalidHandle";// 33
		case hipErrorNotReady: return "hipErrorNotReady";// 34
		case hipErrorInsufficientDriver: return "hipErrorInsufficientDriver";// 35
		case hipErrorSetOnActiveProcess: return "hipErrorSetOnActiveProcess";// 36
		case cudaErrorInvalidSurface: return "cudaErrorInvalidSurface";// 37
		case hipErrorNoDevice: return "hipErrorNoDevice";// 38
		case hipErrorECCNotCorrectable: return "hipErrorECCNotCorrectable";// 39
		case hipErrorSharedObjectSymbolNotFound: return "hipErrorSharedObjectSymbolNotFound";// 40
		case hipErrorSharedObjectInitFailed: return "hipErrorSharedObjectInitFailed";// 41
		case hipErrorUnsupportedLimit: return "hipErrorUnsupportedLimit";// 42
		case cudaErrorDuplicateVariableName: return "cudaErrorDuplicateVariableName";// 43
		case cudaErrorDuplicateTextureName: return "cudaErrorDuplicateTextureName";// 44
		case cudaErrorDuplicateSurfaceName: return "cudaErrorDuplicateSurfaceName";// 45
		case cudaErrorDevicesUnavailable: return "cudaErrorDevicesUnavailable";// 46
		case hipErrorInvalidImage: return "hipErrorInvalidImage";// 47
		case hipErrorNoBinaryForGpu: return "hipErrorNoBinaryForGpu";// 48
		case cudaErrorIncompatibleDriverContext: return "cudaErrorIncompatibleDriverContext";// 49
		case hipErrorPeerAccessAlreadyEnabled: return "hipErrorPeerAccessAlreadyEnabled";// 50
		case hipErrorPeerAccessNotEnabled: return "hipErrorPeerAccessNotEnabled";// 51
		case hipErrorContextAlreadyInUse: return "hipErrorContextAlreadyInUse";// 52
		case hipErrorProfilerDisabled: return "hipErrorProfilerDisabled";// 53
		case hipErrorProfilerNotInitialized: return "hipErrorProfilerNotInitialized";// 54
		case hipErrorProfilerAlreadyStarted: return "hipErrorProfilerAlreadyStarted";// 55
		case hipErrorProfilerAlreadyStopped: return "hipErrorProfilerAlreadyStopped";// 56
		case hipErrorAssert: return "hipErrorAssert";// 57
		case cudaErrorTooManyPeers: return "cudaErrorTooManyPeers";// 58
		case hipErrorHostMemoryAlreadyRegistered: return "hipErrorHostMemoryAlreadyRegistered";// 59
		case hipErrorHostMemoryNotRegistered: return "hipErrorHostMemoryNotRegistered";// 60
		case hipErrorOperatingSystem: return "hipErrorOperatingSystem";// 61
		case cudaErrorStartupFailure: return "cudaErrorStartupFailure";// 62
		case cudaErrorApiFailureBase: return "cudaErrorApiFailureBase";// 63
		default: return "Unknown error";
	}
}


