#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Complex data type
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale(Complex *, const Complex *,
                                                   int, float);

// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int, const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

// The filter size is assumed to be a number smaller than the signal size
#ifndef SIGNAL_SIZE
#define SIGNAL_SIZE (int(80000000))
//#define SIGNAL_SIZE (int(80000000/6))
#endif

#define FILTER_KERNEL_SIZE 8192
#define ITERS 150

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { srand(25890); runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  printf("[simpleCUFFT] is starting...\n");

  findCudaDevice(argc, (const char **)argv);

  // Allocate host memory for the signal
  Complex *h_signal =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < SIGNAL_SIZE; ++i) {
    h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
    h_signal[i].y = 0;
  }

  // Allocate host memory for the filter
  Complex *h_filter_kernel =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * FILTER_KERNEL_SIZE));

  // Initialize the memory for the filter
  for (unsigned int i = 0; i < FILTER_KERNEL_SIZE; ++i) {
    h_filter_kernel[i].x = rand() / static_cast<float>(RAND_MAX);
    h_filter_kernel[i].y = 0;
  }

  // Pad signal and filter kernel
  Complex *h_padded_signal;
  Complex *h_padded_filter_kernel;
  int new_size =
      PadData(h_signal, &h_padded_signal, SIGNAL_SIZE, h_filter_kernel,
              &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
  int mem_size = sizeof(Complex) * new_size;

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&d_signal), mem_size));
  // Copy host memory to device
  memcpy(d_signal, h_padded_signal, mem_size);

  // Allocate device memory for filter kernel
  Complex *d_filter_kernel;
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&d_filter_kernel), mem_size));

    printf("alloced,%ld\n", mem_size * 2);

  // Copy host memory to device
  memcpy(d_filter_kernel, h_padded_filter_kernel, mem_size);

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

  // CUFFT plan advanced API
  hipfftHandle plan_adv;
  size_t workSize;
  long long int new_size_long = new_size;

  checkCudaErrors(hipfftCreate(&plan_adv));
  checkCudaErrors(hipfftXtMakePlanMany(plan_adv, 1, &new_size_long, NULL, 1, 1,
                                      HIP_C_32F, NULL, 1, 1, HIP_C_32F, 1,
                                      &workSize, HIP_C_32F));
  printf("Temporary buffer size %li bytes\n", workSize);

    hipEvent_t start;
    hipEventCreate(&start);

    hipEvent_t stop;
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    for (int i = 0; i < ITERS; i++)
    {
  // Transform signal and kernel
  //printf("Transforming signal hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_FORWARD));
  checkCudaErrors(hipfftExecC2C(
      plan_adv, reinterpret_cast<hipfftComplex *>(d_filter_kernel),
      reinterpret_cast<hipfftComplex *>(d_filter_kernel), HIPFFT_FORWARD));

  // Multiply the coefficients together and normalize the result
  //printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
  ComplexPointwiseMulAndScale<<<32, 256>>>(d_signal, d_filter_kernel, new_size,
                                           1.0f / new_size);
  // Check if kernel execution generated and error
  getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Transform signal back
  //printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_BACKWARD));
}

    // Record the stop event
    hipEventRecord(stop, NULL);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);
    hipDeviceSynchronize();


    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    printf("perf,%lf\n", msecTotal/1000.0);

/*
  // Allocate host memory for the convolution result
  Complex *h_convolved_signal_ref =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));

  printf("Convolve\n");
  // Convolve on the host
  Convolve(h_signal, SIGNAL_SIZE, h_filter_kernel, FILTER_KERNEL_SIZE,
           h_convolved_signal_ref);

  printf("Check\n");
  // check result
  bool bTestResult = sdkCompareL2fe(
      reinterpret_cast<float *>(h_convolved_signal_ref),
      reinterpret_cast<float *>(h_convolved_signal), 2 * SIGNAL_SIZE, 1e-5f);
*/
  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));
  checkCudaErrors(hipfftDestroy(plan_adv));

  // cleanup memory
  free(h_signal);
  free(h_filter_kernel);
  free(h_padded_signal);
  free(h_padded_filter_kernel);
  checkCudaErrors(hipFree(d_signal));
  checkCudaErrors(hipFree(d_filter_kernel));
  
}

// Pad data
int PadData(const Complex *signal, Complex **padded_signal, int signal_size,
            const Complex *filter_kernel, Complex **padded_filter_kernel,
            int filter_kernel_size) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;
  int new_size = signal_size + maxRadius;

  // Pad signal
  Complex *new_data =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * new_size));
  memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
  memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
  *padded_signal = new_data;

  // Pad filter
  new_data = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * new_size));
  memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
  memset(new_data + maxRadius, 0,
         (new_size - filter_kernel_size) * sizeof(Complex));
  memcpy(new_data + new_size - minRadius, filter_kernel,
         minRadius * sizeof(Complex));
  *padded_filter_kernel = new_data;

  return new_size;
}

////////////////////////////////////////////////////////////////////////////////
// Filtering operations
////////////////////////////////////////////////////////////////////////////////

// Computes convolution on the host
void Convolve(const Complex *signal, int signal_size,
              const Complex *filter_kernel, int filter_kernel_size,
              Complex *filtered_signal) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;

  // Loop over output element indices
  for (int i = 0; i < signal_size; ++i) {
    filtered_signal[i].x = filtered_signal[i].y = 0;

    // Loop over convolution indices
    for (int j = -maxRadius + 1; j <= minRadius; ++j) {
      int k = i + j;

      if (k >= 0 && k < signal_size) {
        filtered_signal[i] =
            ComplexAdd(filtered_signal[i],
                       ComplexMul(signal[k], filter_kernel[minRadius - j]));
      }
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b,
                                                   int size, float scale) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = threadID; i < size; i += numThreads) {
    a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
  }
}
