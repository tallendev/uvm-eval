#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <stdlib.h>
#include <chrono>
#include <iostream>
#include "safecuda.h"
#include <assert.h>
#include <random>
// num float in 4k page
#define PSIZE 1024lu //512lu //500 //1024 //512

#ifndef PNUM
#define PNUM  512 //524288 //6000000//6000000 //2097152
#endif
//#define PNUM 262144
#define ARRAY_SIZE (PSIZE * PNUM)

#ifndef THREADS
#define THREADS 64 //128
#endif



#ifndef TASKS_PER_THREAD
#define TASKS_PER_THREAD 100 //1
#endif

#ifndef BLOCKS
#define BLOCKS (1 + PNUM/THREADS) //i(PNUM/TASKS_PER_THREAD/THREADS)  //31250
#endif

//__device__ unsigned long long counter;


static inline void clflush2(volatile void *__p)
{
    asm volatile("clflush (%0)" :: "r" (__p));
}

__device__ unsigned long t1[PNUM];
//__device__ unsigned long t2[PNUM * PSIZE];
__device__ unsigned long tdif[PNUM];
__inline__ __device__ void prefetch_l1 (const void* addr)
{

      asm(" prefetch.global.L1 [ %1 ];": "=l"(addr) : "l"(addr));
}

__inline__ __device__ void prefetch_l2 (const void* addr)
{

      asm(" prefetch.global.L2 [ %1 ];": "=l"(addr) : "l"(addr));
}


static __device__ __inline__ uint64_t __nano(){
  uint64_t mclk;
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(mclk));
  return mclk ;
  }

static __device__ __inline__ uint32_t __myclock(){
  uint32_t mclk;
  asm volatile("mov.u32 %0, %%clock;" : "=r"(mclk));
  return mclk ;}


extern "C"
__global__ void uvmer2(volatile float* a, const __restrict__ size_t* b)
{
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < PNUM)
    {
        float ele = a[b[idx] * PSIZE];
        if (ele == 0.35) 
        {
            //b[idx] = ele;
            a[idx] = b[idx + 73];
        }
    }
}

extern "C"
__global__ void uvmer(volatile float* a, float* b)
{
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < PNUM)
    {
        for (int i = 0; i < TASKS_PER_THREAD; i++)
        {
//            unsigned long timestamp = __nano(); //clock64(); //  __myclock();
            //double ele = a[idx * PSIZE];
//            printf("idx + i: %ld\n", idx + i);
            //double ele = a[(idx + i) * 100  * PSIZE];
            float ele = a[PSIZE * (idx + (i * 10) * (blockDim.x * gridDim.x))];
//            unsigned long timestamp2 = __nano();//clock64(); //__myclock();
            if (ele == 0.35) 
            {
                b[idx] = ele;
                //printf("%lf", ele);
            }
//            t1[idx] = timestamp;
            //t2[idx] = timestamp2;
//            tdif[idx] = timestamp2 - timestamp;
        }
    }
}

extern "C"
__global__ void stupid()
{
    return;
}

void printl(const char* const str)
{
    //printf("######## %s #######\n", str);
}


inline void copyIndexData(size_t* b)
{
    std::vector<size_t> indexes;
    indexes.reserve(PNUM);
    #pragma omp simd
    for (size_t i = 0; i < PNUM; i++)
    {
        //indexes[i] = i;
        indexes.push_back(i);
    }
    std::random_shuffle(indexes.begin(), indexes.end());
    hipMemcpy(b, &indexes[0], PNUM * sizeof(size_t), hipMemcpyHostToDevice);
}

int main(void)
{
    printl("init");
    float* array;
    float* array2;
 //   hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    typedef std::chrono::high_resolution_clock Clock;

    array2=NULL;

    CHECK_CUDA_ERROR();
    stupid<<<1,1>>>();
    hipDeviceSynchronize();
    int khz;
    auto rng = std::default_random_engine {};

    for (int i = 0; i < 1; i++)
    {
        //std::shuffle(indexes.begin(), indexes.end(), rng);
        /*
        for (size_t i = 0; i < (PNUM > 100 ? 100 : PNUM); i++)
        {
            printf("i, indexes[i]: %lu, %lu\n", i, indexes[i]);
        }*/
        size_t* b;
        hipMalloc(&b, PNUM * sizeof(size_t));
        copyIndexData(b);

        CHECK_CUDA_ERROR();
        printf("Allocating %u mb\n", ARRAY_SIZE * sizeof(float) / 100000);
        hipMallocManaged(&array, ARRAY_SIZE * sizeof(float));
        CHECK_CUDA_ERROR();

        printf("alloced,%ld\n", ARRAY_SIZE * sizeof(float));
        #pragma simd
        for (size_t i = 0; i < ARRAY_SIZE; i++) 
        {
            array[i] = 0.0;
        }
        #pragma omp simd
        for (size_t i = 0; i < ARRAY_SIZE; i++) 
        {
            clflush2(array + i);
        }
        hipEvent_t start;
        hipEventCreate(&start);

        hipEvent_t stop;
        hipEventCreate(&stop);

        // Record the start event
        hipEventRecord(start, NULL);

        uvmer2<<<BLOCKS, THREADS>>>(array, b);
        // Record the stop event
        hipEventRecord(stop, NULL);

        // Wait for the stop event to complete
        hipEventSynchronize(stop);
        hipDeviceSynchronize();
        CHECK_CUDA_ERROR();
        float msecTotal = 0.0f;
        hipEventElapsedTime(&msecTotal, start, stop);

        // should be pages / sec
        printf("perf,%lf\n", (BLOCKS * THREADS) / (msecTotal/1000.0));


        
    CHECK_CUDA_ERROR();
        hipFree(array);
        hipFree(b);
    CHECK_CUDA_ERROR();
    }
    CHECK_CUDA_ERROR();
}
