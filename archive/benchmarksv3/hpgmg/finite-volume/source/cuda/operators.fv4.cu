#include "hip/hip_runtime.h"
/*
# Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
#
# Redistribution and use in source and binary forms, with or without
# modification, are permitted provided that the following conditions
# are met:
#  * Redistributions of source code must retain the above copyright
#    notice, this list of conditions and the following disclaimer.
#  * Redistributions in binary form must reproduce the above copyright
#    notice, this list of conditions and the following disclaimer in the
#    documentation and/or other materials provided with the distribution.
#  * Neither the name of NVIDIA CORPORATION nor the names of its
#    contributors may be used to endorse or promote products derived
#    from this software without specific prior written permission.
#
# THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
# EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
# IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
# PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
# CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
# EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
# PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
# PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
# OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
# (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
# OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
//------------------------------------------------------------------------------------------------------------------------------
// Nikolay Sakharnykh
// nsakharnykh@nvidia.com
// Copyright (c) 2014-2015, NVIDIA CORPORATION.  All rights reserved.
//------------------------------------------------------------------------------------------------------------------------------
// Samuel Williams
// SWWilliams@lbl.gov
// Lawrence Berkeley National Lab
//------------------------------------------------------------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
//------------------------------------------------------------------------------------------------------------------------------
#ifdef _OPENMP
#include <omp.h>
#endif
//------------------------------------------------------------------------------------------------------------------------------
#include "../timers.h"
#include "../defines.h"
#include "../level.h"
#include "../operators.h"
//------------------------------------------------------------------------------------------------------------------------------
#define STENCIL_VARIABLE_COEFFICIENT
//------------------------------------------------------------------------------------------------------------------------------
#ifdef STENCIL_FUSE_BC
  #error GPU implementation does not support fusion of the boundary conditions with the operator
#endif
//------------------------------------------------------------------------------------------------------------------------------
#ifdef USE_TEX
  #define       TEX
  #define  X(i)  ( __ldg(&x[i])      )
  #define BI(i)  ( __ldg(&beta_i[i]) )
  #define BJ(i)  ( __ldg(&beta_j[i]) )
  #define BK(i)  ( __ldg(&beta_k[i]) )
#elif  USE_TEXPA
  #define       TEX
  #define  X(i)  ( __ldg(x+i)      )
  #define BI(i)  ( __ldg(beta_i+i) )
  #define BJ(i)  ( __ldg(beta_j+i) )
  #define BK(i)  ( __ldg(beta_k+i) )
#else
  #define  X(i)  ( x[i]      )
  #define BI(i)  ( beta_i[i] )
  #define BJ(i)  ( beta_j[i] )
  #define BK(i)  ( beta_k[i] )
#endif
//------------------------------------------------------------------------------------------------------------------------------
#define Dinv_ijk() Dinv[ijk]        // simply retrieve it rather than recalculating it
//------------------------------------------------------------------------------------------------------------------------------
#define STENCIL_TWELFTH ( 0.0833333333333333333)  // 1.0/12.0;
//------------------------------------------------------------------------------------------------------------------------------
#ifdef STENCIL_VARIABLE_COEFFICIENT
  #ifdef USE_HELMHOLTZ // Helmholtz
  #define H0    ( a*alpha[ijk]*X(ijk) )
  #else // Poisson
  #define H0
  #endif
  #define apply_op_ijk()                                                                                                           	\
  (																	\
  H0 - b*h2inv*(															\
  STENCIL_TWELFTH*(															\
  + BI(ijk        )*( 15.0*(X(ijk-1      )-X(ijk)) - (X(ijk-2        )-X(ijk+1      )) )						\
  + BI(ijk+1      )*( 15.0*(X(ijk+1      )-X(ijk)) - (X(ijk+2        )-X(ijk-1      )) )						\
  + BJ(ijk        )*( 15.0*(X(ijk-jStride)-X(ijk)) - (X(ijk-2*jStride)-X(ijk+jStride)) )						\
  + BJ(ijk+jStride)*( 15.0*(X(ijk+jStride)-X(ijk)) - (X(ijk+2*jStride)-X(ijk-jStride)) )						\
  + BK(ijk        )*( 15.0*(X(ijk-kStride)-X(ijk)) - (X(ijk-2*kStride)-X(ijk+kStride)) )						\
  + BK(ijk+kStride)*( 15.0*(X(ijk+kStride)-X(ijk)) - (X(ijk+2*kStride)-X(ijk-kStride)) ) )						\
																	\
  + 0.25*STENCIL_TWELFTH*(                                                                                                              \
  + (BI(ijk        +jStride)-BI(ijk        -jStride)) * (X(ijk-1      +jStride)-X(ijk+jStride)-X(ijk-1      -jStride)+X(ijk-jStride))	\
  + (BI(ijk        +kStride)-BI(ijk        -kStride)) * (X(ijk-1      +kStride)-X(ijk+kStride)-X(ijk-1      -kStride)+X(ijk-kStride))	\
  + (BJ(ijk        +1      )-BJ(ijk        -1      )) * (X(ijk-jStride+1      )-X(ijk+1      )-X(ijk-jStride-1      )+X(ijk-1      ))	\
  + (BJ(ijk        +kStride)-BJ(ijk        -kStride)) * (X(ijk-jStride+kStride)-X(ijk+kStride)-X(ijk-jStride-kStride)+X(ijk-kStride))	\
  + (BK(ijk        +1      )-BK(ijk        -1      )) * (X(ijk-kStride+1      )-X(ijk+1      )-X(ijk-kStride-1      )+X(ijk-1      ))	\
  + (BK(ijk        +jStride)-BK(ijk        -jStride)) * (X(ijk-kStride+jStride)-X(ijk+jStride)-X(ijk-kStride-jStride)+X(ijk-jStride))	\
																	\
  + (BI(ijk+1      +jStride)-BI(ijk+1      -jStride)) * (X(ijk+1      +jStride)-X(ijk+jStride)-X(ijk+1      -jStride)+X(ijk-jStride))	\
  + (BI(ijk+1      +kStride)-BI(ijk+1      -kStride)) * (X(ijk+1      +kStride)-X(ijk+kStride)-X(ijk+1      -kStride)+X(ijk-kStride))	\
  + (BJ(ijk+jStride+1      )-BJ(ijk+jStride-1      )) * (X(ijk+jStride+1      )-X(ijk+1      )-X(ijk+jStride-1      )+X(ijk-1      ))	\
  + (BJ(ijk+jStride+kStride)-BJ(ijk+jStride-kStride)) * (X(ijk+jStride+kStride)-X(ijk+kStride)-X(ijk+jStride-kStride)+X(ijk-kStride))	\
  + (BK(ijk+kStride+1      )-BK(ijk+kStride-1      )) * (X(ijk+kStride+1      )-X(ijk+1      )-X(ijk+kStride-1      )+X(ijk-1      ))	\
  + (BK(ijk+kStride+jStride)-BK(ijk+kStride-jStride)) * (X(ijk+kStride+jStride)-X(ijk+jStride)-X(ijk+kStride-jStride)+X(ijk-jStride)) )	\
  )																	\
  )
#else // constant coefficient
  #define apply_op_ijk()	 	\
  (					\
  a*x[ijk] - b*h2inv*STENCIL_TWELFTH*(	\
  - 1.0*(X(ijk-2*kStride) +		\
         X(ijk-2*jStride) +		\
         X(ijk-2        ) +		\
         X(ijk+2        ) +		\
         X(ijk+2*jStride) +		\
         X(ijk+2*kStride) )		\
  +16.0*(X(ijk  -kStride) +		\
         X(ijk  -jStride) +		\
         X(ijk  -1      ) +		\
         X(ijk  +1      ) +		\
         X(ijk  +jStride) +		\
         X(ijk  +kStride) )		\
  -90.0*(X(ijk          ) ) 		\
  )					\
  )
#endif
//------------------------------------------------------------------------------------------------------------------------------
#ifdef  USE_GSRB
#define GSRB_OOP
#define NUM_SMOOTHS      3 // RBRBRB
#elif   USE_CHEBY
#define NUM_SMOOTHS      1
#define CHEBYSHEV_DEGREE 6 // i.e. one degree-6 polynomial smoother
#elif   USE_JACOBI
#define NUM_SMOOTHS      6
#elif   USE_L1JACOBI
#define NUM_SMOOTHS      6
#else
#error You must compile CUDA code with either -DUSE_GSRB, -DUSE_CHEBY, -DUSE_JACOBI, -DUSE_L1JACOBI, or -DUSE_SYMGS
#endif
//------------------------------------------------------------------------------------------------------------------------------
// include smoother
#include "extra.h"
#if defined(USE_GSRB) && ( defined(GSRB_STRIDE2) || defined(GSRB_BRANCH) || (defined(GSRB_FP)&&!defined(GSRB_OOP)) )
  #include "stencils/gsrb.h"
#else
  #ifdef USE_SHM // shared memory
  #include "stencils/smooth.smem.fv4.h"
  #elif  USE_REG // registers
  #include "stencils/smooth.reg.fv4.h"
  #else // baseline
  #include "stencils/smooth.base.h"
  #endif
#endif
//------------------------------------------------------------------------------------------------------------------------------
// include residual
#ifdef USE_SHM // shared memory
#include "stencils/residual.reg.fv4.h"
#elif  USE_REG // registers
#include "stencils/residual.reg.fv4.h"
#else // baseline
#include "stencils/residual.base.h"
#endif
//------------------------------------------------------------------------------------------------------------------------------
// include other kernels
#include "blockCopy.h"
#include "misc.h"
#include "boundary_fv.h"
#include "restriction.h"
#include "interpolation_v2.h"
#include "interpolation_v4.h"
//------------------------------------------------------------------------------------------------------------------------------
